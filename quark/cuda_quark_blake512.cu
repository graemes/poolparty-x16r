#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <memory.h>
#include <sys/types.h> // off_t

#include "miner.h"
#include "cuda_helper.h"

#define ROTR(x,n) ROTR64(x,n)

// use sp kernel on SM 5+
#define SP_KERNEL

#define USE_SHUFFLE 0

__constant__
static uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

// ---------------------------- BEGIN CUDA quark_blake512 functions ------------------------------------

__device__ __constant__
static const uint8_t c_sigma_big[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },

	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },

	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
static const uint64_t c_u512[16] =
{
	0x243f6a8885a308d3ULL, 0x13198a2e03707344ULL,
	0xa4093822299f31d0ULL, 0x082efa98ec4e6c89ULL,
	0x452821e638d01377ULL, 0xbe5466cf34e90c6cULL,
	0xc0ac29b7c97c50ddULL, 0x3f84d5b5b5470917ULL,
	0x9216d5d98979fb1bULL, 0xd1310ba698dfb5acULL,
	0x2ffd72dbd01adfb7ULL, 0xb8e1afed6a267e96ULL,
	0xba7c9045f12c7f99ULL, 0x24a19947b3916cf7ULL,
	0x0801f2e2858efc16ULL, 0x636920d871574e69ULL
};

#define G(a,b,c,d,x) { \
	uint32_t idx1 = sigma[i][x]; \
	uint32_t idx2 = sigma[i][x+1]; \
	v[a] += (m[idx1] ^ u512[idx2]) + v[b]; \
	v[d] = SWAPDWORDS(v[d] ^ v[a]); \
	v[c] += v[d]; \
	v[b] = ROTR( v[b] ^ v[c], 25); \
	v[a] += (m[idx2] ^ u512[idx1]) + v[b]; \
	v[d] = ROTR( v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR( v[b] ^ v[c], 11); \
}

__device__ __forceinline__
void quark_blake512_compress(uint64_t *h, const uint64_t *block, const uint8_t ((*sigma)[16]), const uint64_t *u512, const int T0)
{
	uint64_t v[16];
	uint64_t m[16];

	#pragma unroll
	for(int i=0; i < 16; i++) {
		m[i] = cuda_swab64(block[i]);
	}

	//#pragma unroll 8
	for(int i=0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = u512[0];
	v[ 9] = u512[1];
	v[10] = u512[2];
	v[11] = u512[3];
	v[12] = u512[4] ^ T0;
	v[13] = u512[5] ^ T0;
	v[14] = u512[6];
	v[15] = u512[7];

	//#pragma unroll 16
	for(int i=0; i < 16; i++)
	{
		/* column step */
		G( 0, 4, 8, 12, 0 );
		G( 1, 5, 9, 13, 2 );
		G( 2, 6, 10, 14, 4 );
		G( 3, 7, 11, 15, 6 );
		/* diagonal step */
		G( 0, 5, 10, 15, 8 );
		G( 1, 6, 11, 12, 10 );
		G( 2, 7, 8, 13, 12 );
		G( 3, 4, 9, 14, 14 );
	}

	h[0] ^= v[0] ^ v[8];
	h[1] ^= v[1] ^ v[9];
	h[2] ^= v[2] ^ v[10];
	h[3] ^= v[3] ^ v[11];
	h[4] ^= v[4] ^ v[12];
	h[5] ^= v[5] ^ v[13];
	h[6] ^= v[6] ^ v[14];
	h[7] ^= v[7] ^ v[15];
}

__global__ __launch_bounds__(256, 4)
void quark_blake512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *g_nonceVector, uint64_t *g_hash)
{
}

__global__ __launch_bounds__(256,4)
void quark_blake512_gpu_hash_80(uint32_t threads, uint32_t startNounce, void *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t buf[16];
		#pragma unroll
		for (int i=0; i < 16; ++i)
			buf[i] = c_PaddedMessage80[i];

		// The test Nonce
		const uint32_t nounce = startNounce + thread;
		((uint32_t*)buf)[19] = cuda_swab32(nounce);

		uint64_t h[8] = {
			0x6a09e667f3bcc908ULL,
			0xbb67ae8584caa73bULL,
			0x3c6ef372fe94f82bULL,
			0xa54ff53a5f1d36f1ULL,
			0x510e527fade682d1ULL,
			0x9b05688c2b3e6c1fULL,
			0x1f83d9abfb41bd6bULL,
			0x5be0cd19137e2179ULL
		};

		quark_blake512_compress(h, buf, c_sigma_big, c_u512, 640);

		uint64_t *outHash = (uint64_t*)outputHash + (thread * 8U);
		for (uint32_t i=0; i < 8; i++) {
			outHash[i] = cuda_swab64( h[i] );
		}
	}
}

#include "cuda_quark_blake512_sp.cuh"

__host__
void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order)
{
	quark_blake512_cpu_hash_64_sp(threads, startNounce, d_nonceVector, d_outputHash);
	
	//MyStreamSynchronize(NULL, order, thr_id);
}

__host__
void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash)
{
	quark_blake512_cpu_hash_80_sp(threads, startNounce, d_outputHash);
}

// ---------------------------- END CUDA quark_blake512 functions ------------------------------------

__host__
void quark_blake512_cpu_init(int thr_id, uint32_t threads)
{
	cuda_get_arch(thr_id);
}

__host__
void quark_blake512_cpu_free(int thr_id)
{
}

// ----------------------------- Host midstate for 80-bytes input ------------------------------------

#undef SPH_C32
#undef SPH_T32
#undef SPH_C64
#undef SPH_T64

extern "C" {
#include "sph/sph_blake.h"
}

__host__
void quark_blake512_cpu_setBlock_80(int thr_id, uint32_t *endiandata)
{
	quark_blake512_cpu_setBlock_80_sp(thr_id, (uint64_t*) endiandata);
	
	CUDA_LOG_ERROR();
}
